#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <algorithm> 
#include <math.h>
#include <time.h>
#include <fstream>
#include <iostream>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <>
#define Infinity 65536
#define index(x,y,z) (z+y*x)
#define CUDA_ERROR_CHECK
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )
inline void __cudaSafeCall(hipError_t err, const char *file, const int line)
{
#ifdef CUDA_ERROR_CHECK
	if (hipSuccess != err)
	{
		fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n",
			file, line, hipGetErrorString(err));
		exit(-1);
	}
#endif

	return;
}

inline void __cudaCheckError(const char *file, const int line)
{
#ifdef CUDA_ERROR_CHECK
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n",
			file, line, hipGetErrorString(err));
		exit(-1);
	}
	err = hipDeviceSynchronize();
	if (hipSuccess != err)
	{
		fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
			file, line, hipGetErrorString(err));
		exit(-1);
	}
#endif

	return;
}
using namespace std;
const double initial_pheromone = 1.0;
const double evap_rate = 0.5;
const double ALFA = 1;
const double BETA = 2;
int *load_adjacency_matrix(char const *filename, int &n_cities);
int calculate_tourcost(int *distances, int *path, int n_cities);
int *optimal_solution(int *tours, int *distances, int n_ants, int n_cities);
void evaporate(double *pheromones, int n_cities);
void pheromone_update(double *pheromones, int *distances, int*min_path, int n_cities);
int *aco_cuda(int *distances, int n_cities, int n_ants, int minimum_cost);
__global__ void cuda_evaporate(double *pheromones, int n_cities, double evap_rate);
__global__ void cuda_pheromone_update(double *pheromones, int *distances, int *path, int n_cities, double amount);
__global__ void cuda_path_traverse(int *tours, int *visited, double *choiceinfo, double *probs, int n_cities);

int main()
{
	srand((unsigned)time(NULL));
	char const *inputfile, *outputfile;
	inputfile = "Matrix.txt";
	outputfile = "Output.txt";
	int n_cities;
	int *distances;
	distances = load_adjacency_matrix(inputfile, n_cities);
	int min_path;
/** FOR BRUTE FORCE
vector<int> vertex;
	int s = 0;
	for (int i = 0; i < n_cities; i++)
		if (i != s)
			vertex.push_back(i);
	min_path = INT_MAX;
	do {
		int pathweight = 0;
		int k = s;
		for (int i = 0; i < vertex.size(); i++) {
			pathweight += distances[index(n_cities, k, vertex[i])];
			k = vertex[i];
		}
		pathweight += distances[index(n_cities, k, s)];
		min_path = min(min_path,pathweight);
	} while (next_permutation(vertex.begin(), vertex.end()));**/
	ifstream adj_matrix;
	adj_matrix.open("cost.txt");
	adj_matrix >> min_path;
	cout << "Minimum Cost by brute Force: " << min_path << endl;
	int *solution = aco_cuda(distances, n_cities, n_cities,min_path);
	int cost = calculate_tourcost(distances, solution, n_cities);
	ofstream output;
	output.open(outputfile);
	output << "Total cost of traversal: " << cost << endl;
	output << "Best Solution Path:\n";
	for (int i = 0; i < n_cities; i++)
		output << solution[i] << endl;
	output << solution[0] << endl;
	cout << "CUDA ACO is complete" << endl;
	return 0;
}

__global__ void cuda_evaporate(double *pheromones, int n_cities, double evap_rate)
{
	int edge_id = threadIdx.x + blockIdx.x*blockDim.x;
	pheromones[edge_id] *= evap_rate;
}

__global__ void cuda_pheromone_update(double *pheromones, int *distances, int *path, int n_cities, double amount)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	int first = path[id];
	int second = path[id + 1];
	pheromones[index(n_cities, first, second)] += amount;
	pheromones[index(n_cities, second, first)] += amount;
}

__global__ void cuda_path_traverse(int *tours, int *visited, double *choiceinfo, double *probs, int n_cities)
{
	int line_id = blockDim.x*blockIdx.x + threadIdx.x;
	for (int step = 1; step < n_cities; step++)
	{
		int current = tours[index(n_cities, line_id, step - 1)];
		double total_prob = 0.0;
		for (int i = 0; i < n_cities; i++)
		{
			if (visited[index(n_cities, line_id, i)] == 1)
				probs[index(n_cities, line_id, i)] = 0.0;
			else {
				double current_prob = choiceinfo[index(n_cities, current, i)];
				probs[index(n_cities, line_id, i)] = current_prob;
				total_prob += current_prob;
			}
		}
		double random;
		hiprandState_t state;
		hiprand_init((unsigned long long) clock(), 0, 0, &state);
		random = hiprand_uniform(&state);
		random *= total_prob;
		int next;
		double sum = probs[index(n_cities, line_id, 0)];
		for (next = 0; sum < random; next++)
		{
			sum += probs[index(n_cities, line_id, next + 1)];
		}
		tours[index(n_cities, line_id, step)] = next;
		visited[index(n_cities, line_id, next)] = 1;
	}
}

int *load_adjacency_matrix(char const *filename, int &n_cities)
{
	ifstream adj_matrix;
	adj_matrix.open(filename);
	adj_matrix >> n_cities;
	int* distances = (int *)malloc(n_cities*n_cities * sizeof(int));
	for (int i = 0; i < n_cities; i++)
		for (int j = 0; j < n_cities; j++)
			adj_matrix >> distances[index(n_cities, i, j)];
	return distances;
}

int calculate_tourcost(int *distances, int *path, int n_cities)
{
	int cost = 0;
	for (int i = 0; i < (n_cities - 1); i++)
		cost += distances[index(n_cities, path[i], path[i + 1])];
	cost += distances[index(n_cities, path[n_cities-1], path[0])];
	return cost;
}

int *optimal_solution(int *tours, int *distances, int n_ants, int n_cities)
{
	int *best_tour = &tours[0];
	for (int tour = 0; tour < n_ants; tour++)
		if (calculate_tourcost(distances, &tours[index(n_cities, tour, 0)], n_cities) < calculate_tourcost(distances, best_tour, n_cities))
			best_tour = &tours[index(n_cities, tour, 0)];
	return best_tour;
}

void evaporate(double *pheromones, int n_cities)
{
	int size = n_cities * n_cities * sizeof(double);
	double *pheromones_device;
	CudaSafeCall(hipMalloc((void**)&pheromones_device, size));
	hipMemcpy(pheromones_device, pheromones, size, hipMemcpyHostToDevice);
	cuda_evaporate << < n_cities, n_cities >> > (pheromones_device, n_cities, evap_rate);
	CudaCheckError();
	hipMemcpy(pheromones, pheromones_device, size, hipMemcpyDeviceToHost);
	hipFree(pheromones_device);
}

void pheromone_update(double *pheromones, int *distances, int *path, int n_cities)
{
	double amount = (double)(1.0f / (double)calculate_tourcost(distances, path, n_cities));
	int size_path = n_cities * sizeof(int);
	int size_int = n_cities * n_cities * sizeof(int);
	int size_double = n_cities * n_cities * sizeof(double);
	int *path_device;
	int *distances_device;
	double *pheromones_device;
	CudaSafeCall(hipMalloc((void**)&path_device, size_path));
	CudaSafeCall(hipMalloc((void**)&distances_device, size_int));
	CudaSafeCall(hipMalloc((void**)&pheromones_device, size_double));
	hipMemcpy(path_device, path, size_path, hipMemcpyHostToDevice);
	hipMemcpy(distances_device, distances, size_int, hipMemcpyHostToDevice);
	hipMemcpy(pheromones_device, pheromones, size_double, hipMemcpyHostToDevice);
	cuda_pheromone_update << < 1, n_cities - 1 >> > (pheromones_device, distances_device, path_device, n_cities, amount);
	CudaCheckError();
	hipMemcpy(distances, distances_device, size_int, hipMemcpyDeviceToHost);
	hipMemcpy(pheromones, pheromones_device, size_double, hipMemcpyDeviceToHost);
	hipFree(path_device);
	hipFree(distances_device);
	hipFree(pheromones_device);
}

int *aco_cuda(int *distances, int n_cities, int n_ants,int minimum_cost)
{
	int ph_size = n_cities * n_cities * sizeof(double);
	int tours_size = n_ants * n_cities * sizeof(int);
	int dist_size = n_cities * n_cities * sizeof(int);
	double *pheromones = (double*)malloc(ph_size);
	int *tours = (int*)malloc(tours_size);
	int *visited = (int*)malloc(tours_size);
	double *choiceinfo = (double*)malloc(ph_size);
	int *distances_device;
	int *tours_device;
	int *visited_device;
	double *choiceinfo_device;
	double *probs;
	CudaSafeCall(hipMalloc((void**)&distances_device, dist_size));
	CudaSafeCall(hipMalloc((void**)&tours_device, tours_size));
	CudaSafeCall(hipMalloc((void**)&visited_device, tours_size));
	CudaSafeCall(hipMalloc((void**)&choiceinfo_device, ph_size));
	CudaSafeCall(hipMalloc((void**)&probs, ph_size));
	hipMemcpy(distances_device, distances, dist_size, hipMemcpyHostToDevice);
	for (int i = 0; i < n_cities; i++)
		for (int j = 0; j < n_cities; j++)
			pheromones[index(n_cities, i, j)] = initial_pheromone;
	int iteration = 0;
	int best_cost = 0;
	int flag = 0;
	int answer_iteration;
	int valid;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	while(true)
	{
		iteration++;
		best_cost = 0;
		for (int i = 0; i < n_ants; i++)
			for (int j = 0; j < n_cities; j++)
				tours[index(n_cities, i, j)] = Infinity;

		for (int i = 0; i < n_ants; i++)
			for (int j = 0; j < n_cities; j++)
				visited[index(n_cities, i, j)] = 0;

		for (int i = 0; i < n_cities; i++)
		{
			for (int j = 0; j < n_cities; j++)
			{
				double edge_pherom = pheromones[index(n_cities, i, j)];
				double edge_weight = distances[index(n_cities, i, j)];
				double prob = 0.0f;
				if (edge_weight != 0.0f)
				{
					prob = pow(edge_pherom, ALFA)*pow((1 / edge_weight), BETA);
				}
				else
				{
					prob = pow(edge_pherom, ALFA)*pow(Infinity, BETA);
				}
				choiceinfo[index(n_cities, i, j)] = prob;
			}
		}
		hipMemcpy(choiceinfo_device, choiceinfo, ph_size, hipMemcpyHostToDevice);
		for (int ant = 0; ant < n_ants; ant++)
		{
			int step = 0;
			int init = rand() % n_cities;
			tours[index(n_cities, ant, step)] = init;
			visited[index(n_cities, ant, init)] = 1;
		}
		hipMemcpy(visited_device, visited, tours_size, hipMemcpyHostToDevice);
		hipMemcpy(tours_device, tours, tours_size, hipMemcpyHostToDevice);
		cuda_path_traverse <<< 1, n_ants >>> (tours_device, visited_device, choiceinfo_device, probs, n_cities);
		CudaCheckError();
		hipMemcpy(tours, tours_device, tours_size, hipMemcpyDeviceToHost);
		hipMemcpy(visited, visited_device, tours_size, hipMemcpyDeviceToHost);
		evaporate(pheromones, n_cities);
		int *best = optimal_solution(tours, distances, n_ants, n_cities);
		best_cost = calculate_tourcost(distances, best, n_cities);
		cout << "Iteration: " << iteration <<"\t"<< "Best cost in iteration: "<<best_cost<< endl;
		if (best_cost == minimum_cost)
		{
			if (flag == 0)
			{
				hipEventRecord(stop);
				answer_iteration = iteration;
				flag = 1;
				valid = 100;
			}
			else
			{
				if (valid == 0)
				{
					break;
				}
				else
				{
					valid--;
				}
			}
		}
		else
		{
			flag = 0;
			valid = 100;
		}



		pheromone_update(pheromones, distances, best, n_cities);
	}
	cout << "Iteration in which we got answer" << answer_iteration << endl;
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	cout << "Time" << milliseconds << "ms" << endl;
	hipFree(distances_device);
	hipFree(tours_device);
	hipFree(visited_device);
	hipFree(choiceinfo_device);
	hipFree(probs);
	int *best = optimal_solution(tours, distances, n_ants, n_cities);
	return best;
}